#include <hip/hip_runtime.h>
#include <stdlib.h>
#define N 4096 // size of array
int main(int argc, char *argv[])
{
hipEvent_t start, stop; // using cuda events to measure time
float elapsed_time_ms;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start, 0 ); // instrument code to measure start time
vectorAdd<<<B,T>>>(dev_a,dev_b,dev_c);
hipMemcpy(c,dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 ); // instrument code to measure end time
    hipEventSynchronize( stop);
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    printf("Time to calculate results: %f ms.\n", elapsed_time_ms);
}