#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void sm(char* a,char* c,int n,int dup){
    int i = threadIdx.x;
    for(int j=0;j<dup;j++){
        c[i+j*n] = a[i];
    }
}

int main(void){
    
    char *d_a,*d_c;
    int n,dup;
    int size = sizeof(char);
    char a[200],c[200];
    printf("Enter string \n");
    scanf("%s",a);
    n = strlen(a);
    printf("Enter number of duplication \n");
    scanf("%d",&dup);
    hipMalloc((void**)&d_a,(size*n*dup));
    hipMalloc((void**)&d_c,(size*n*dup));
    hipMemcpy(d_a,&a,(size*n),hipMemcpyHostToDevice);
    hipEvent_t start, stop; // using cuda events to measure time
float elapsed_time_ms;
hipEventCreate( &start );
hipEventCreate( &stop );
    sm<<<1,n>>>(d_a,d_c,n,dup);
    hipMemcpy(&c,d_c,(size*n*dup),hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 ); // instrument code to measure end time
hipEventSynchronize( stop);
hipEventElapsedTime( &elapsed_time_ms, start, stop );
printf("Time to calculate results: %f ms.\n", elapsed_time_ms);
    printf("Resultant array \n");
    for (int i = 0; i <(n*dup); i++)
        printf("%c ",c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_c);
    return 0;    
}