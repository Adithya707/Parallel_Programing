#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void wr(char* a,char* c,int n){
    int i = threadIdx.x;
    if(c[i]!=32){
       c[i] = a[n-i-1];
    }
    else{
        c[i] = 32;
    }
    
}

int main(void){
    
    char *d_a,*d_c,*d_b;
    int n;
    int size = sizeof(char);
    char a[20],c[20];
    printf("Enter string \n");
    scanf("%s",a);
    n = strlen(a);
    hipMalloc((void**)&d_a,(size*n));
    hipMalloc((void**)&d_c,(size*n));
    hipMalloc((void**)&d_b,(size*n));
    hipMemcpy(d_a,&a,(size*n),hipMemcpyHostToDevice);
    hipEvent_t start, stop; // using cuda events to measure time
float elapsed_time_ms;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start, 0 ); // instrument code to measure start time
    wr<<<1,n>>>(d_a,d_c,n);
    hipMemcpy(&c,d_c,(size*n),hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 ); // instrument code to measure end time
hipEventSynchronize( stop);
hipEventElapsedTime( &elapsed_time_ms, start, stop );
printf("Time to calculate results: %f ms.\n", elapsed_time_ms);
    printf("Resultant array \n");
    for (int i = 0; i < n; i++)
        printf("%c ",c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_c);
    return 0;

    
}