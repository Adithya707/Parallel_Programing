#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void st(char* a,char* c,int n){
    int i = threadIdx.x;
    if (a[i]>='A' && a[i]<='Z')
        c[i] = a[i] + 'a' - 'A';
    else if (a[i]>='a' && a[i]<='z')
        c[i] = a[i] + 'A' - 'a';
}

int main(void){
    
    char *d_a,*d_c;
    int n;
    int size = sizeof(char);
    char a[20],c[20];
    printf("Enter string \n");
    scanf("%s",a);
    n = strlen(a);
    hipMalloc((void**)&d_a,(size*n));
    hipMalloc((void**)&d_c,(size*n));
    hipMemcpy(d_a,&a,(size*n),hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float elapsed_time_ms;
hipEventCreate( &start );
hipEventCreate( &stop );
hipEventRecord( start, 0 );
    st<<<1,n>>>(d_a,d_c,n);
    hipMemcpy(&c,d_c,(size*n),hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 ); // instrument code to measure end time
hipEventSynchronize( stop);
hipEventElapsedTime( &elapsed_time_ms, start, stop );
printf("Time to calculate results: %f ms.\n", elapsed_time_ms);
    printf("Resultant array \n");
    for (int i = 0; i < n; i++)
        printf("%c ",c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_c);
    return 0;
}