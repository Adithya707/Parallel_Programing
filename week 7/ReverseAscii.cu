#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void Rev(char* a,char* c,int n){
    int i = threadIdx.x;
    int b = a[i];
    int r = 0;
    while (b > 0) {
        int digit = b % 10;
        r = r * 10 + digit;
        b /= 10;
    }
    c[i] = r;
}

int main(void){
    
    char *d_a,*d_c;
    int n;
    int size = sizeof(char);
    char a[20],c[20];
    printf("Enter string \n");
    scanf("%s",a);
    n = strlen(a);
    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipMalloc((void**)&d_a,(size*n));
    hipMalloc((void**)&d_c,(size*n));
    hipMemcpy(d_a,&a,(size*n),hipMemcpyHostToDevice);
    hipEventRecord( start, 0 ); // instrument code to measure start time
    Rev<<<1,n>>>(d_a,d_c,n);
    hipMemcpy(&c,d_c,(size*n),hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 ); // instrument code to measure end time
hipEventSynchronize( stop);
hipEventElapsedTime( &elapsed_time_ms, start, stop );
printf("Time to calculate results: %f ms.\n", elapsed_time_ms);
    printf("Resultant array \n");
    for (int i = 0; i < n; i++)
        printf("%c ",c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_c);
    return 0;

    
}