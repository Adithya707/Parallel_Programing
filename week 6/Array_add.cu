#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{

    int *d_a, *d_b, *d_c, n;
    int size = sizeof(int);
    printf("Enter size of array \n");
    scanf("%d", &n);
    int a[n * 2], b[n * 2], c[n * 2], i;
    hipMalloc((void **)&d_a, (size * n));
    hipMalloc((void **)&d_b, (size * n));
    hipMalloc((void **)&d_c, (size * n));
    printf("Enter value for a and b\n");
    for (i = 0; i < n; i++)
    {
        printf("Enter valur for a[%d] ", i);
        scanf("%d", &a[i]);
        printf("Enter valur for b[%d] ", i);
        scanf("%d", &b[i]);
    }
    printf("\n");
    hipMemcpy(d_a, &a, (size * n), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, (size * n), hipMemcpyHostToDevice);
    add<<<1, n>>>(d_a, d_b, d_c, n);
    hipMemcpy(&c, d_c, (size * n), hipMemcpyDeviceToHost);
    printf("Resultant array \n");
    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}