#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x;
    int decimal = a[i];
    int octal = 0, k = 1;
    while (decimal != 0)
    {
        octal += (decimal % 8) * k;
        decimal /= 8;
        k *= 10;
    }
    c[i] = octal;
}

int main(void)
{

    int *d_a, *d_b, *d_c, n;
    int size = sizeof(int);

    printf("Enter size of array \n");
    scanf("%d", &n);
    int a[n * 2], b[n * 2], c[n * 2], i;
    hipMalloc((void **)&d_a, (size * n));
    hipMalloc((void **)&d_b, (size * n));
    hipMalloc((void **)&d_c, (size * n));
    printf("Enter value for a and b\n");
    for (i = 0; i < n; i++)
    {
        printf("Enter valur for a[%d] ", i);
        scanf("%d", &a[i]);
    }
    printf("\n");
    hipMemcpy(d_a, &a, (size * n), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, (size * n), hipMemcpyHostToDevice);
    add<<<1, n>>>(d_a, d_b, d_c, n);
    hipMemcpy(&c, d_c, (size * n), hipMemcpyDeviceToHost);
    printf("Resultant array \n");
    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}