#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>

#include<hip/hip_runtime.h>

__global__ void sum(int * A, int * B, int * C, int n) {
  int idx = blockIdx.y * blockDim.y + threadIdx.y; 
  int idy = blockIdx.x * blockDim.x + threadIdx.x;
  int row = 0;
  int col = 0;
  if (idx < n && idy < n) {
    for(int i=0;i<n;i++){
        row += A[idx * n + i]; 
        col += A[i * n + idy];
    }
    B[idx] += row;
    C[idy] += col;
  }
}

int main() {
  int b[100], c[100], i;
  int * d_a, * d_b, * d_c;
  int size = sizeof(int);
  int n = 3;
  hipMalloc((void ** ) & d_a, size * n * n);
  hipMalloc((void ** ) & d_b, size * n);
  hipMalloc((void ** ) & d_c, size * n);
  printf("Enter array elements \n");
  int a[9] = {
    1,
    2,
    3,
    4,
    5,
    6,
    7,
    8,
    9
  };
  hipMemcpy(d_a, & a, size * n * n, hipMemcpyHostToDevice);
  dim3 dimBlock(n, n);
  dim3 dimGrid(1, 1);
  sum << < dimGrid, dimBlock >>> (d_a, d_b, d_c, n);
  hipMemcpy( & b, d_b, size * n, hipMemcpyDeviceToHost);
  hipMemcpy( & c, d_c, size * n, hipMemcpyDeviceToHost);
  for (i = 0; i < n; i++) {
    printf("Sum of row %d is %d \n", i + 1, b[i]);
    printf("Sum of column %d is %d \n", i + 1, c[i]);
  }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
